#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
 * This is a program of password cracking of two alphabets and four digits using cuda. 
 * AV72, FT27, IR75, SC55 these passwords were provided and adding some digits to make four digits.
 * 

 To Compile:
    nvcc -o CudaCrack CudaCrack.cu 
  
  To run:
    ./CudaCrack 
     *****************************************************************************/



__device__ char password_text[4][7]={"AV7212","ES2112","GT5912","RB9612"};


__device__ void displayResult(char *password)
{
 printf("Password found is: %s\n",password);
}

__device__ void is_a_match(char *perform) {
  char *a = perform;
  char *b = perform;
  char *c = perform;
  char *d = perform;
  char *ps_1 = password_text[0];
  char *ps_2 = password_text[1];
  char *ps_3 = password_text[2];
  char *ps_4 = password_text[3];


  while(*a == *ps_1) { 
   if(*a == '\0') 
    {
        displayResult(perform);
      break;
    }

    a++;
    ps_1++;
  }
	
  while(*b == *ps_2) { 
   if(*b == '\0') 
    {
        displayResult(perform);
      break;
    }

    b++;
    ps_2++;
  }


  while(*c == *ps_3) { 
   if(*c == '\0') 
    {
	displayResult(perform);
      break;
    }

    c++;
    ps_3++;
  }


  while(*d == *ps_4) { 
   if(*d == '\0') 
    {
       displayResult(perform);
      break;
    }

    d++;
    ps_4++;
  }
  return;

}


__global__ void  kernel() {
char w,x,y,z;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char first_val = i; 
char second_val = j; 
    
password[0] = first_val;
password[1] = second_val;

	for(w='0'; w<='9'; w++){
	  for(x='0'; x<='9'; x++){
	   for(y='0'; y<='9'; y++){
	     for(z='0'; z<='9'; z++){
	        password[2] = w;
	        password[3] = x;
	        password[4] = y;
	        password[5] = z; 
	      is_a_match(password); 
	   }
	}
	}
	}

}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

   dim3 block_Dim(26,1,1), thread_Dim(26,1,1);
  kernel <<<block_Dim,thread_Dim>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


